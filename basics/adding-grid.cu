// First, I just have to turn our add function into a function that the GPU can run, called a kernel in CUDA. To do this, all I have to do is add the specifier __global__ to the function, which tells the CUDA C++ compiler that this is a function that runs on the GPU and can be called from CPU code.


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays

// Find the number of CUDA cores per SM
 // Your RTX 3050 has 16 SMs, each with 128 CUDA cores, giving you 2048 CUDA cores total.

__global__ void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1000000;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Optimal configuration
  int blockSize = 1024; // Max threads per block
  int numBlocks = (N + blockSize - 1) / blockSize;

  // Run kernel on 1M elements on the GPU
  add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}